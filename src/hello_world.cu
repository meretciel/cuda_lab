
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU!\n");
}


int main() {
    printf("Hello World from CPU!\n");
    helloFromGPU <<<1, 10>>>();
    hipDeviceReset();
    return 0;
}

